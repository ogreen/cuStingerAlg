#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <inttypes.h>

#include <math.h>


#include "utils.hpp"
#include "update.hpp"
#include "cuStinger.hpp"

#include "static_k_truss/k_truss.cuh"	
using namespace cuStingerAlgs;

#define CUDA(call, ...) do {                        \
        hipError_t _e = (call);                    \
        if (_e == hipSuccess) break;               \
        fprintf(stdout,                             \
                "CUDA runtime error: %s (%d)\n",    \
                hipGetErrorString(_e), _e);        \
        return -1;                                  \
    } while (0)


#define STAND_PRINTF(sys, time, triangles) printf("%s : \t%ld \t%f\n", sys,triangles, time);


// int arrayBlocks[]={16000};
// int arrayBlockSize[]={32,64,96,128,192,256};
// int arrayThreadPerIntersection[]={1,2,4,8,16,32};
// int arrayThreadShift[]={0,1,2,3,4,5};
// int arrayBlocks[]={64000};
// int arrayBlockSize[]={256};
// int arrayThreadPerIntersection[]={32};
// int arrayThreadShift[]={5};

int arrayBlocks[]={16000};
int arrayBlockSize[]={32};
int arrayThreadPerIntersection[]={1};
int arrayThreadShift[]={0};


// int arrayBlocks[]={64000};
// int arrayBlockSize[]={64};
// int arrayThreadPerIntersection[]={8};
// int arrayThreadShift[]={3};

int runKtruss(vertexId_t nv,length_t ne, int alg, int maxk, length_t*  off,vertexId_t*  ind,string graphName)
{
	int device = 0;
	int run    = 2;
//  int scriptMode =atoi(argv[PAR_SCRIPT]);
//	int sps =atoi(argv[PAR_SP]);	
//	int tsp =atoi(argv[PAR_T_SP]);	
//	int nbl =atoi(argv[PAR_NUM_BL]);
//	int shifter =atoi(argv[PAR_SHIFT]);
		
	struct hipDeviceProp_t prop;
	hipGetDeviceProperties	(&prop,device);	
    length_t *d_off = NULL;	
    vertexId_t* d_ind = NULL;
	triangle_t *d_triangles = NULL;  

   	int * triNE = (int *) malloc ((ne ) * sizeof (int));	
	int64_t allTrianglesCPU=0;
	
		hipSetDevice(device);
		CUDA(hipMalloc(&d_off, sizeof(length_t)*(nv+1)));
		CUDA(hipMalloc(&d_ind, sizeof(vertexId_t)*ne));
		CUDA(hipMalloc(&d_triangles, sizeof(triangle_t)*(nv+1)));

		CUDA(hipMemcpy(d_off, off, sizeof(length_t)*(nv+1), hipMemcpyHostToDevice));
		CUDA(hipMemcpy(d_ind, ind, sizeof(vertexId_t)*ne, hipMemcpyHostToDevice));

		triangle_t* h_triangles = (triangle_t *) malloc ( sizeof(triangle_t)*(nv+1)  );		

		float minTime=10e9,time,minTimecuStinger=10e9;

		int64_t sumDevice=0;

		int blocksToTest=sizeof(arrayBlocks)/sizeof(int);
		int blockSizeToTest=sizeof(arrayBlockSize)/sizeof(int);
		int tSPToTest=sizeof(arrayThreadPerIntersection)/sizeof(int);
		for(int b=0;b<blocksToTest; b++){
		    int blocks=arrayBlocks[b];
			for(int bs=0; bs<blockSizeToTest; bs++){
			    int sps=arrayBlockSize[bs];
			    for(int t=0; t<tSPToTest;t++){
		            int tsp=arrayThreadPerIntersection[t];
					int shifter=arrayThreadShift[t];
					int nbl=sps/tsp;

					cuStinger custing(defaultInitAllocater,defaultUpdateAllocater);

					cuStingerInitConfig hipInit;
					hipInit.initState =eInitStateCSR;
					hipInit.maxNV = nv+1;
					hipInit.useVWeight = false;hipInit.isSemantic = false; hipInit.useEWeight = true;
					hipInit.csrNV 			= nv;	hipInit.csrNE	   		= ne;
					hipInit.csrOff 			= off;	hipInit.csrAdj 			= ind;
					hipInit.csrVW 			= NULL;	hipInit.csrEW			= NULL;

					hipEvent_t ce_start,ce_stop;
					float totalTime;

					if(alg&1){
						custing.initializeCuStinger(hipInit);
						kTruss kt;
						kt.setInitParameters(nv,ne, tsp,nbl,shifter,blocks, sps);
						kt.Init(custing);
						kt.copyOffsetArrayDevice(d_off);
						kt.Reset();
						start_clock(ce_start, ce_stop);
						
						kt.Run(custing);
						totalTime = end_clock(ce_start, ce_stop);
						cout << "graph="<< graphName<< endl; 
						cout << "k=" << kt.getMaxK() << ":" << totalTime << endl; 
						kt.Release();
						custing.freecuStinger();

					}
					if(alg&2){
						cuStinger custing2(defaultInitAllocater,defaultUpdateAllocater);
						custing2.initializeCuStinger(hipInit);

						kTruss kt2;
						kt2.setInitParameters(nv,ne, tsp,nbl,shifter,blocks, sps);
						kt2.Init(custing2);
						kt2.copyOffsetArrayDevice(d_off);
						kt2.Reset();
						start_clock(ce_start, ce_stop);
						
						kt2.RunDynamic(custing2);

						totalTime = end_clock(ce_start, ce_stop);
						cout << "graph="<< graphName<< endl; 
						cout << "k=" << kt2.getMaxK() << ":" << totalTime << endl; 
						kt2.Release();

						if(totalTime<minTimecuStinger) minTimecuStinger=totalTime; 

						custing2.freecuStinger();
					}
					if(alg&4){
						custing.initializeCuStinger(hipInit);
						kTruss kt;
						kt.setInitParameters(nv,ne, tsp,nbl,shifter,blocks, sps);
						kt.Init(custing);
						kt.copyOffsetArrayDevice(d_off);
						kt.Reset();
						start_clock(ce_start, ce_stop);
						
						kt.RunForK(custing,maxk);
						totalTime = end_clock(ce_start, ce_stop);
						cout << "graph="<< graphName<< endl; 
						cout << "k=" << kt.getMaxK() << ":" << totalTime << endl; 
						kt.Release();
						custing.freecuStinger();

					}


			    }
			}	
		}
		free(h_triangles);

		CUDA(hipFree(d_off));
		CUDA(hipFree(d_ind));
		CUDA(hipFree(d_triangles));
	// }
	free(triNE);
    return 0;
}



int main(const int argc, char *argv[]){
	int device=0;
    hipSetDevice(device);
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, device);
 
    length_t nv, ne,*off;
    vertexId_t *adj;

	bool isDimacs,isSNAP,isRmat=false,isMarket;
	string filename(argv[1]);
	isDimacs = filename.find(".graph")==std::string::npos?false:true;
	isSNAP   = filename.find(".txt")==std::string::npos?false:true;
	isRmat 	 = filename.find("kron")==std::string::npos?false:true;
	isMarket = filename.find(".mtx")==std::string::npos?false:true;

	string graphName=filename;
	int period = filename.find(".");

	int i, lasti=0;
	for(i = graphName.find("/", 0); i != std::string::npos; i = graphName.find("/", i)){
		i++;
		lasti=i;
	}
	graphName = graphName.substr(lasti,period-lasti);

	// cout << lasti << " " << period << endl;
	// cout << graphName << endl;

	if(isDimacs){
	    readGraphDIMACS(argv[1],&off,&adj,&nv,&ne,isRmat);
	}
	else if(isSNAP){
	    readGraphSNAP(argv[1],&off,&adj,&nv,&ne,true);
	}
	else if(isMarket){
		readGraphMatrixMarket(argv[1],&off,&adj,&nv,&ne,(isRmat)?false:true);
	}
	else{ 
		cout << "Unknown graph type" << endl;
	}
	int alg=3,maxk=3;
	if (argc>=3)
		alg = atoi(argv[2]);
	if (argc>=4)
		maxk = atoi(argv[3]);

	cout << "Vertices: " << nv << "    Edges: " << ne  << "      " << off[nv] << endl;

	hipEvent_t ce_start,ce_stop;

	runKtruss(nv,ne,alg,maxk,off,adj,graphName);



	free(off);
	free(adj);
    return 0;	
}
