#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <inttypes.h>

#include <math.h>


#include "utils.hpp"
#include "update.hpp"
#include "cuStinger.hpp"

// #include "static_triangle_counting/cct.hpp"

#include "static_k_truss/k_truss.cuh"	
using namespace cuStingerAlgs;


#define CUDA(call, ...) do {                        \
        hipError_t _e = (call);                    \
        if (_e == hipSuccess) break;               \
        fprintf(stdout,                             \
                "CUDA runtime error: %s (%d)\n",    \
                hipGetErrorString(_e), _e);        \
        return -1;                                  \
    } while (0)



#define STAND_PRINTF(sys, time, triangles) printf("%s : \t%ld \t%f\n", sys,triangles, time);


// int arrayBlocks[]={16000};
// int arrayBlockSize[]={32,64,96,128,192,256};
// int arrayThreadPerIntersection[]={1,2,4,8,16,32};
// int arrayThreadShift[]={0,1,2,3,4,5};
// int arrayBlocks[]={64000};
// int arrayBlockSize[]={256};
// int arrayThreadPerIntersection[]={32};
// int arrayThreadShift[]={5};

int arrayBlocks[]={16000};
int arrayBlockSize[]={192};
int arrayThreadPerIntersection[]={8};
int arrayThreadShift[]={3};


void initHostTriangleArray(triangle_t* h_triangles, vertexId_t nv){	
	for(vertexId_t sd=0; sd<(nv);sd++){
		h_triangles[sd]=0;
	}
}

int64_t sumTriangleArray(triangle_t* h_triangles, vertexId_t nv){	
	int64_t sum=0;
	for(vertexId_t sd=0; sd<(nv);sd++){
	  sum+=h_triangles[sd];
	}
	return sum;
}

int comparecuStingerAndCSR(cuStinger& custing, vertexId_t nv,length_t ne, int maxK, length_t*  off,vertexId_t*  ind)
{
	int device = 0;
	int run    = 2;
//  int scriptMode =atoi(argv[PAR_SCRIPT]);
//	int sps =atoi(argv[PAR_SP]);	
//	int tsp =atoi(argv[PAR_T_SP]);	
//	int nbl =atoi(argv[PAR_NUM_BL]);
//	int shifter =atoi(argv[PAR_SHIFT]);
		
	struct hipDeviceProp_t prop;
	hipGetDeviceProperties	(&prop,device);	
    length_t *d_off = NULL;	
    vertexId_t* d_ind = NULL;
	triangle_t *d_triangles = NULL;  

   	int * triNE = (int *) malloc ((ne ) * sizeof (int));	
	int64_t allTrianglesCPU=0;
	
	// if(run&1){
	// 	hipEvent_t startCPU, stopCPU;
	// 	float timeCPU;
	// 	hipEventCreate(&startCPU); hipEventCreate(&stopCPU);
	// 	hipEventRecord(startCPU, 0);
	// 	hostCountTriangles (nv, off,ind, triNE, &allTrianglesCPU);
	// 	hipEventRecord(stopCPU, 0);hipEventSynchronize(stopCPU);
		
	// 	hipDeviceSynchronize();hipEventElapsedTime(&timeCPU, startCPU, stopCPU);
	// 	// STAND_PRINTF("CPU", timeCPU,allTrianglesCPU)
	// }	

	// if(run&2){
		hipSetDevice(device);
		CUDA(hipMalloc(&d_off, sizeof(length_t)*(nv+1)));
		CUDA(hipMalloc(&d_ind, sizeof(vertexId_t)*ne));
		CUDA(hipMalloc(&d_triangles, sizeof(triangle_t)*(nv+1)));

		CUDA(hipMemcpy(d_off, off, sizeof(length_t)*(nv+1), hipMemcpyHostToDevice));
		CUDA(hipMemcpy(d_ind, ind, sizeof(vertexId_t)*ne, hipMemcpyHostToDevice));

		triangle_t* h_triangles = (triangle_t *) malloc ( sizeof(triangle_t)*(nv+1)  );		

		float minTime=10e9,time,minTimecuStinger=10e9;

		int64_t sumDevice=0;
		initHostTriangleArray(h_triangles,nv);

		int blocksToTest=sizeof(arrayBlocks)/sizeof(int);
		int blockSizeToTest=sizeof(arrayBlockSize)/sizeof(int);
		int tSPToTest=sizeof(arrayThreadPerIntersection)/sizeof(int);
		for(int b=0;b<blocksToTest; b++){
		    int blocks=arrayBlocks[b];
			for(int bs=0; bs<blockSizeToTest; bs++){
			    int sps=arrayBlockSize[bs];
			    for(int t=0; t<tSPToTest;t++){
		            int tsp=arrayThreadPerIntersection[t];
					int shifter=arrayThreadShift[t];
					int nbl=sps/tsp;

				cuStinger custing(defaultInitAllocater,defaultUpdateAllocater);

				cuStingerInitConfig hipInit;
				hipInit.initState =eInitStateCSR;
				hipInit.maxNV = nv+1;
				hipInit.useVWeight = false;hipInit.isSemantic = false; hipInit.useEWeight = false;
				hipInit.csrNV 			= nv;	hipInit.csrNE	   		= ne;
				hipInit.csrOff 			= off;	hipInit.csrAdj 			= ind;
				hipInit.csrVW 			= NULL;	hipInit.csrEW			= NULL;

				custing.initializeCuStinger(hipInit);

					hipEvent_t ce_start,ce_stop;
					float totalTime;

					kTruss kt;
					kt.setInitParameters(nv,ne, tsp,nbl,shifter,blocks, sps);
					kt.Init(custing);
					kt.copyOffsetArrayDevice(d_off);
					kt.Reset();
					start_clock(ce_start, ce_stop);
					
					if(maxK==-1)
						kt.Run(custing);
					else
						kt.RunForK(custing,maxK);


						// KTrussOneIteration(custing, d_triangles, tsp,nbl,shifter,blocks, sps);

					totalTime = end_clock(ce_start, ce_stop);
					cout << "Total time for k-Truss = " << kt.getMaxK() << " : " << totalTime << endl; 
					kt.Release();

					if(totalTime<minTimecuStinger) minTimecuStinger=totalTime; 

					custing.freecuStinger();


					// CUDA(hipMemcpy(d_triangles, h_triangles, sizeof(triangle_t)*(nv+1), hipMemcpyHostToDevice));
					// start_clock(ce_start, ce_stop);
					// 	KTrussOneIteration(custing, d_triangles, tsp,nbl,shifter,blocks, sps);
					// time = end_clock(ce_start, ce_stop);
					// CUDA(hipMemcpy(h_triangles, d_triangles, sizeof(triangle_t)*(nv+1), hipMemcpyDeviceToHost));

					// if(time<minTimecuStinger) minTimecuStinger=time; 
					// sumDevice=sumTriangleArray(h_triangles,nv);initHostTriangleArray(h_triangles,nv);

					// printf("### %d %d %d %d %d \t\t %ld \t %f\n", blocks,sps, tsp, nbl, shifter,sumDevice, time);
			    }
			}	
		}
		// STAND_PRINTF("GPU - csr     ", minTime,sumDevice)
		// STAND_PRINTF("GPU - custing ", minTimecuStinger,sumDevice)
		// cout << "Vertices " << nv << endl;
		// cout << "Edges " << ne << endl;

		cout << nv << ", " << ne << ", "<< minTime << ", " << minTimecuStinger<< endl;

		free(h_triangles);

		CUDA(hipFree(d_off));
		CUDA(hipFree(d_ind));
		CUDA(hipFree(d_triangles));
	// }
	free(triNE);
    return 0;
}



int main(const int argc, char *argv[]){
	int device=0;
    hipSetDevice(device);
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, device);
 
    length_t nv, ne,*off;
    vertexId_t *adj;

	bool isDimacs,isSNAP,isRmat=false,isMarket;
	string filename(argv[1]);
	isDimacs = filename.find(".graph")==std::string::npos?false:true;
	isSNAP   = filename.find(".txt")==std::string::npos?false:true;
	isRmat 	 = filename.find("kron")==std::string::npos?false:true;
	isMarket = filename.find(".mtx")==std::string::npos?false:true;

	if(isDimacs){
	    readGraphDIMACS(argv[1],&off,&adj,&nv,&ne,isRmat);
	}
	else if(isSNAP){
	    readGraphSNAP(argv[1],&off,&adj,&nv,&ne,true);
	}
	else if(isMarket){
		readGraphMatrixMarket(argv[1],&off,&adj,&nv,&ne,(isRmat)?false:true);
	}
	else{ 
		cout << "Unknown graph type" << endl;
	}
	int maxK=-1;
	if (argc==3)
		maxK = atoi(argv[2]);

	cout << "Vertices: " << nv << "    Edges: " << ne  << "      " << off[nv] << endl;

	hipEvent_t ce_start,ce_stop;
	cuStinger custing(defaultInitAllocater,defaultUpdateAllocater);

	cuStingerInitConfig hipInit;
	hipInit.initState =eInitStateCSR;
	hipInit.maxNV = nv+1;
	hipInit.useVWeight = false;
	hipInit.isSemantic = false;  // Use edge types and vertex types
	hipInit.useEWeight = false;
	// CSR data
	hipInit.csrNV 			= nv;
	hipInit.csrNE	   		= ne;
	hipInit.csrOff 			= off;
	hipInit.csrAdj 			= adj;
	hipInit.csrVW 			= NULL;
	hipInit.csrEW			= NULL;

	custing.initializeCuStinger(hipInit);

	comparecuStingerAndCSR(custing,nv,ne,maxK,off,adj);

	custing.freecuStinger();
	cout << "Vertices: " << nv << "    Edges: " << ne  << endl;


	free(off);
	free(adj);
    return 0;	
}
