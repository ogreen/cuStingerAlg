#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <inttypes.h>

#include <math.h>

#include "update.hpp"
#include "cuStinger.hpp"

#include "operators.cuh"

#include "static_k_truss/k_truss.cuh"

using namespace cuStingerAlgs;


__device__ void initialize(const vertexId_t diag_id, const length_t u_len, length_t v_len,
    length_t* const __restrict__ u_min, length_t* const __restrict__ u_max,
    length_t* const __restrict__ v_min, length_t* const __restrict__ v_max,
    int* const __restrict__ found)
{
	if (diag_id == 0){
		*u_min=*u_max=*v_min=*v_max=0;
		*found=1;
	}
	else if (diag_id < u_len){
		*u_min=0; *u_max=diag_id;
		*v_max=diag_id;*v_min=0;
	}
	else if (diag_id < v_len){
		*u_min=0; *u_max=u_len;
		*v_max=diag_id;*v_min=diag_id-u_len;
	}
	else{
		*u_min=diag_id-v_len; *u_max=u_len;
		*v_min=diag_id-u_len; *v_max=v_len;
	}
}

__device__ void workPerThread(const length_t uLength, const length_t vLength, 
	const int threadsPerIntersection, const int threadId,
    int * const __restrict__ outWorkPerThread, int * const __restrict__ outDiagonalId){
  int totalWork = uLength + vLength;
  int remainderWork = totalWork%threadsPerIntersection;
  int workPerThread = totalWork/threadsPerIntersection;

  int longDiagonals  = (threadId > remainderWork) ? remainderWork:threadId;
  int shortDiagonals = (threadId > remainderWork) ? (threadId - remainderWork):0;

  *outDiagonalId = ((workPerThread+1)*longDiagonals) + (workPerThread*shortDiagonals);
  *outWorkPerThread = workPerThread + (threadId < remainderWork);
}

__device__ void bSearch(unsigned int found, const vertexId_t diagonalId,
    vertexId_t const * const __restrict__ uNodes, vertexId_t const * const __restrict__ vNodes,
    length_t const * const __restrict__ uLength, 
    length_t * const __restrict__ outUMin, length_t * const __restrict__ outUMax,
    length_t * const __restrict__ outVMin, length_t * const __restrict__ outVMax,    
    length_t * const __restrict__ outUCurr,
    length_t * const __restrict__ outVCurr){
  	length_t length;
	
	while(!found) {
	    *outUCurr = (*outUMin + *outUMax)>>1;
	    *outVCurr = diagonalId - *outUCurr;
	    if(*outVCurr >= *outVMax){
			length = *outUMax - *outUMin;
			if(length == 1){
				found = 1;
				continue;
			}
	    }

	    unsigned int comp1 = uNodes[*outUCurr] > vNodes[*outVCurr-1];
	    unsigned int comp2 = uNodes[*outUCurr-1] > vNodes[*outVCurr];
	    if(comp1 && !comp2){
			found = 1;
	    }
	    else if(comp1){
	      *outVMin = *outVCurr;
	      *outUMax = *outUCurr;
	    }
	    else{
	      *outVMax = *outVCurr;
	      *outUMin = *outUCurr;
	    }
  	}

	if((*outVCurr >= *outVMax) && (length == 1) && (*outVCurr > 0) &&
	(*outUCurr > 0) && (*outUCurr < (*uLength - 1))){
		unsigned int comp1 = uNodes[*outUCurr] > vNodes[*outVCurr - 1];
		unsigned int comp2 = uNodes[*outUCurr - 1] > vNodes[*outVCurr];
		if(!comp1 && !comp2){(*outUCurr)++; (*outVCurr)--;}
	}
}


__device__ int fixStartPoint(const length_t uLength, const length_t vLength,
    length_t * const __restrict__ uCurr, length_t * const __restrict__ vCurr,
    vertexId_t const * const __restrict__ uNodes, vertexId_t const * const __restrict__ vNodes){
	
	unsigned int uBigger = (*uCurr > 0) && (*vCurr < vLength) && (uNodes[*uCurr-1] == vNodes[*vCurr]);
	unsigned int vBigger = (*vCurr > 0) && (*uCurr < uLength) && (vNodes[*vCurr-1] == uNodes[*uCurr]);
	*uCurr += vBigger;
	*vCurr += uBigger;
	return (uBigger + vBigger);
}

__device__ vertexId_t* binSearch(vertexId_t *a, vertexId_t x, length_t n)
{
	length_t min = 0, max = n, acurr, curr;// = (min+max)/2
	do {
		curr = (min+max)/2;
		acurr = a[curr];
		min = (x > acurr) ? curr : min;
		max = (x < acurr) ? curr : max;
	} while (x != acurr || min != max);
	return a + curr;
}

__device__ int32_t findIndexOfVertex(cuStinger* custing,vertexId_t src,vertexId_t dst__){
	length_t srcLen=custing->dVD->used[src];
	vertexId_t* adj_src=custing->dVD->adj[src]->dst;
	for(vertexId_t adj=0; adj<srcLen; adj+=1){
		vertexId_t dst = adj_src[adj];
		if(dst==dst__)
			return adj;
	}
	// printf("This should never happpen\n");
	return -1;
}

__device__ void indexBinarySearch(vertexId_t* data, length_t arrLen, vertexId_t key, int& pos){
	int low = 0;
	int high = arrLen-1;
	while(high >= low) {
	     int middle = (low + high) / 2;
	     if(data[middle] == key) {
			 pos=middle;
	         return;
	     }
	     if(data[middle] < key) {
	         low = middle + 1;
	     }
	     if(data[middle] > key) {
	         high = middle - 1;
	     }
	}
	return;		
}

__device__ void findIndexOfTwoVerticesBinary(cuStinger* custing,vertexId_t src,vertexId_t v1, vertexId_t v2,
			int &pos_v1, int &pos_v2){
	vertexId_t* adj_src=custing->dVD->adj[src]->dst;
	length_t srcLen=custing->dVD->used[src];
	pos_v1=-1;
	pos_v2=-1;

	indexBinarySearch(adj_src, srcLen, v1,pos_v1);
	indexBinarySearch(adj_src, srcLen, v2,pos_v2);
}

__device__ void findIndexOfTwoVertices(cuStinger* custing,vertexId_t src,vertexId_t v1, vertexId_t v2,
			int &pos_v1, int &pos_v2){
	length_t srcLen=custing->dVD->used[src];
	vertexId_t* adj_src=custing->dVD->adj[src]->dst;

	pos_v1=-1;
	pos_v2=-1;
	for(vertexId_t adj=0; adj<srcLen; adj+=1){
		vertexId_t dst = adj_src[adj];
		if(dst==v1)
			pos_v1=adj;
		if(dst==v2)
			pos_v2=adj;
		if(pos_v1!=-1 && pos_v2!=-1)
			return;
	}
	printf("This should never happpen\n");
	return;
}


template <bool uMasked, bool vMasked, bool subtract, bool upd3rdV>
__device__ void intersectCount(cuStinger* custing,const length_t uLength, const length_t vLength,
    vertexId_t const * const __restrict__ uNodes, vertexId_t const * const __restrict__ vNodes,
    length_t * const __restrict__ uCurr, length_t * const __restrict__ vCurr,
    int * const __restrict__ workIndex, int * const __restrict__ workPerThread,
    int * const __restrict__ triangles, int found, triangle_t * const __restrict__ outPutTriangles, 
    vertexId_t const * const __restrict__ uMask, vertexId_t const * const __restrict__ vMask, triangle_t multiplier, 
	vertexId_t src, vertexId_t dest,vertexId_t u, vertexId_t v)
{
  if((*uCurr < uLength) && (*vCurr < vLength)){
    int comp;
    int vmask;
    int umask;
    while(*workIndex < *workPerThread){
    	vmask = (vMasked) ? vMask[*vCurr] : 0;
        umask = (uMasked) ? uMask[*uCurr] : 0;
		comp = uNodes[*uCurr] - vNodes[*vCurr];

		*triangles += (comp == 0 && !umask && !vmask);

		if (upd3rdV && comp == 0 && !umask && !vmask)
			if (subtract) {
				// atomicSub(outPutTriangles + uNodes[*uCurr], multiplier);

				// Ktruss
				vertexId_t common = uNodes[*uCurr];
				length_t pos_id;

					length_t posu,posv;
					findIndexOfTwoVerticesBinary(custing,common,u,v,posu,posv);
					
					if(posu!=-1)
						atomicSub(custing->dVD->adj[common]->ew+posu,1);
					else
						printf("1");

					if(posv!=-1)
						atomicSub(custing->dVD->adj[common]->ew+posv,1);
					else
						printf("2");

						atomicSub(custing->dVD->adj[u]->ew+*uCurr,1);
						atomicSub(custing->dVD->adj[v]->ew+*vCurr,1);


			}
			// else {
				// 	atomicAdd(outPutTriangles + uNodes[*uCurr], multiplier);
			// }
		*uCurr += (comp <= 0 && !vmask) || umask;
		*vCurr += (comp >= 0 && !umask) || vmask;
		*workIndex += (comp == 0&& !umask && !vmask) + 1;

		if((*vCurr == vLength) || (*uCurr == uLength)){
			break;
		}
    }
    *triangles -= ((comp == 0) && (*workIndex > *workPerThread) && (found));
  }
}



// u_len < v_len
template <bool uMasked, bool vMasked, bool subtract, bool upd3rdV>
__device__ triangle_t count_triangles(cuStinger* custing,vertexId_t u, vertexId_t const * const __restrict__ u_nodes, length_t u_len,
    vertexId_t v, vertexId_t const * const __restrict__ v_nodes, length_t v_len, int threads_per_block,
    volatile vertexId_t* __restrict__ firstFound, int tId, triangle_t * const __restrict__ outPutTriangles,
    vertexId_t const * const __restrict__ uMask, vertexId_t const * const __restrict__ vMask, triangle_t multiplier, 
	vertexId_t src, vertexId_t dest)
{

	// Partitioning the work to the multiple thread of a single GPU processor. The threads should get a near equal number of the elements to Tersect - this number will be off by 1.
	int work_per_thread, diag_id;
	workPerThread(u_len, v_len, threads_per_block, tId, &work_per_thread, &diag_id);
	triangle_t triangles = 0;
	int work_index = 0,found=0;
	length_t u_min,u_max,v_min,v_max,u_curr,v_curr;

	firstFound[tId]=0;

	if(work_per_thread>0){
		// For the binary search, we are figuring out the initial poT of search.
		initialize(diag_id, u_len, v_len,&u_min, &u_max,&v_min, &v_max,&found);
    	u_curr = 0; v_curr = 0;

	    bSearch(found, diag_id, u_nodes, v_nodes, &u_len, &u_min, &u_max, &v_min,
        &v_max, &u_curr, &v_curr);

    	int sum = fixStartPoint(u_len, v_len, &u_curr, &v_curr, u_nodes, v_nodes);
    	work_index += sum;
	    if(tId > 0)
	      firstFound[tId-1] = sum;
	    triangles += sum;
	    intersectCount<uMasked, vMasked, subtract, upd3rdV>(custing,
	    	u_len, v_len, u_nodes, v_nodes, &u_curr, &v_curr,
	        &work_index, &work_per_thread, &triangles, firstFound[tId], outPutTriangles, 
	        uMask, vMask, multiplier,src,dest,u,v);
	}
	return triangles;
}


__device__ void workPerBlock(const length_t numVertices,
    length_t * const __restrict__ outMpStart,
    length_t * const __restrict__ outMpEnd, int blockSize)
{
	length_t verticesPerMp = numVertices/gridDim.x;
	length_t remainderBlocks = numVertices % gridDim.x;
	length_t extraVertexBlocks = (blockIdx.x > remainderBlocks)? remainderBlocks:blockIdx.x;
	length_t regularVertexBlocks = (blockIdx.x > remainderBlocks)? blockIdx.x - remainderBlocks:0;

	length_t mpStart = ((verticesPerMp+1)*extraVertexBlocks) + (verticesPerMp*regularVertexBlocks);
	*outMpStart = mpStart;
	*outMpEnd = mpStart + verticesPerMp + (blockIdx.x < remainderBlocks);
}


__global__ void devicecuStingerKTruss(cuStinger* custing,
    triangle_t * const __restrict__ outPutTriangles, const int threads_per_block,
    const int number_blocks, const int shifter,kTrussData* devData)
{
	vertexId_t nv = custing->nv;
	// Partitioning the work to the multiple thread of a single GPU processor. The threads should get a near equal number of the elements to intersect - this number will be off by no more than one.
	int tx = threadIdx.x;
 	vertexId_t this_mp_start, this_mp_stop;

	const int blockSize = blockDim.x;
	workPerBlock(nv, &this_mp_start, &this_mp_stop, blockSize);

	__shared__ triangle_t  s_triangles[1024];
	__shared__ vertexId_t firstFound[1024];

	length_t adj_offset=tx>>shifter;
	length_t* firstFoundPos=firstFound + (adj_offset<<shifter);
	for (vertexId_t src = this_mp_start; src < this_mp_stop; src++){
		length_t srcLen=custing->dVD->getUsed()[src];
	    triangle_t tCount = 0;	    
		for(int k=adj_offset; k<srcLen; k+=number_blocks){
			vertexId_t dest = custing->dVD->getAdj()[src]->dst[k];
			int destLen=custing->dVD->getUsed()[dest];

			 if (dest<src) 
			 	continue;

			bool avoidCalc = (src == dest) || (destLen < 2) || (srcLen < 2);
			if(avoidCalc)
				continue;

	        bool sourceSmaller = (srcLen<destLen);
	        vertexId_t small = sourceSmaller? src : dest;
	        vertexId_t large = sourceSmaller? dest : src;
	        length_t small_len = sourceSmaller? srcLen : destLen;
	        length_t large_len = sourceSmaller? destLen : srcLen;

	        const vertexId_t* small_ptr = custing->dVD->getAdj()[small]->dst;
	        const vertexId_t* large_ptr = custing->dVD->getAdj()[large]->dst;
	        // triangle_t triFound = count_triangles<false,false,false,true>
	        triangle_t triFound = count_triangles<false,false,false,false>
						(custing,small, small_ptr, small_len,
						 large,large_ptr, large_len,
						 threads_per_block,firstFoundPos,
						 tx%threads_per_block, outPutTriangles, NULL, NULL,1,src,dest);
	        tCount +=triFound; 
	        int pos=devData->offsetArray[src]+k;
	        atomicAdd(devData->trianglePerEdge+pos,triFound);
            pos=-1;
			indexBinarySearch(custing->dVD->getAdj()[dest]->dst, destLen, src,pos);

	        pos=devData->offsetArray[dest]+pos;
	        atomicAdd(devData->trianglePerEdge+pos,triFound);
		}
	//	s_triangles[tx] = tCount;
	//	blockReduce(&outPutTriangles[src],s_triangles,blockSize);
	}
}

void KTrussOneIteration(cuStinger& custing,
    triangle_t * const __restrict__ outPutTriangles, const int threads_per_block,
    const int number_blocks, const int shifter, const int thread_blocks, const int blockdim, kTrussData* devData){

	devicecuStingerKTruss<<<thread_blocks, blockdim>>>(custing.devicePtr(), outPutTriangles, threads_per_block,number_blocks,shifter,devData);
}


__global__ void devicecuStingerNewTriangles(cuStinger* custing, BatchUpdateData *bud,
    triangle_t * const __restrict__ outPutTriangles, const int threads_per_block,
    const int number_blocks, const int shifter, bool deletion,
    length_t const * const __restrict__ redCU)
{
	length_t batchSize = *(bud->getBatchSize());
	// Partitioning the work to the multiple thread of a single GPU processor. The threads should get a near equal number of the elements to intersect - this number will be off by no more than one.
	int tx = threadIdx.x;
 	length_t this_mp_start, this_mp_stop;

	// length_t *d_off = bud->getOffsets();
	vertexId_t * d_ind = bud->getDst();
	vertexId_t * d_seg = bud->getSrc();

	const int blockSize = blockDim.x;
	workPerBlock(batchSize, &this_mp_start, &this_mp_stop, blockSize);

	__shared__ vertexId_t firstFound[1024];

	length_t adj_offset=tx>>shifter;
	length_t* firstFoundPos=firstFound + (adj_offset<<shifter);
	for (length_t edge = this_mp_start+adj_offset; edge < this_mp_stop; edge+=number_blocks){
		if (bud->getIndDuplicate()[edge]==1) {// this means it's a duplicate edge
			continue;
		}
		vertexId_t src = d_seg[edge];
		vertexId_t dest= d_ind[edge];

		if (src < dest) continue;

		length_t srcLen=custing->dVD->getUsed()[src];
		length_t destLen=custing->dVD->getUsed()[dest];		

		bool avoidCalc = ((src == dest) || (destLen == 0) || (srcLen == 0));
		if(avoidCalc)
			continue;

		bool sourceSmaller = (srcLen<destLen);
        vertexId_t small = sourceSmaller? src : dest;
        vertexId_t large = sourceSmaller? dest : src;
        length_t small_len = sourceSmaller? srcLen : destLen;
        length_t large_len = sourceSmaller? destLen : srcLen;
		if(small_len==0)
			printf("hello oded\n");

        const vertexId_t* small_ptr = custing->dVD->getAdj()[small]->dst;
	    const vertexId_t* large_ptr = custing->dVD->getAdj()[large]->dst;

		triangle_t tCount = count_triangles<false,false,true,true>(
								custing,small, small_ptr, small_len,
								large,large_ptr, large_len,
								threads_per_block,firstFoundPos,
								tx%threads_per_block, outPutTriangles,
								NULL, NULL, 2,src,dest);
		__syncthreads();
	}
}


template <bool uMasked, bool vMasked, bool subtract, bool upd3rdV>
__device__ void intersectCountAsymmetric(cuStinger* custing,const length_t uLength, const length_t vLength,
    vertexId_t const * const __restrict__ uNodes, vertexId_t const * const __restrict__ vNodes,
    length_t * const __restrict__ uCurr, length_t * const __restrict__ vCurr,
    int * const __restrict__ workIndex, int * const __restrict__ workPerThread,
    int * const __restrict__ triangles, int found, triangle_t * const __restrict__ outPutTriangles, 
    vertexId_t const * const __restrict__ uMask, vertexId_t const * const __restrict__ vMask, triangle_t multiplier, 
	vertexId_t src, vertexId_t dest,vertexId_t u, vertexId_t v)
{


  if((*uCurr < uLength) && (*vCurr < vLength)){
    int comp;
    int vmask;
    int umask;
    while(*workIndex < *workPerThread){
    	vmask = (vMasked) ? vMask[*vCurr] : 0;
        umask = (uMasked) ? uMask[*uCurr] : 0;
		comp = uNodes[*uCurr] - vNodes[*vCurr];

		*triangles += (comp == 0 && !umask && !vmask);

		if (upd3rdV && comp == 0 && !umask && !vmask)
			if (subtract) {
				atomicSub(outPutTriangles + uNodes[*uCurr], multiplier);

				// Ktruss
				length_t pos_id;
				vertexId_t common = uNodes[*uCurr];

				if(dest==u)
					atomicSub(custing->dVD->adj[dest]->ew+*uCurr,1);
				else
					atomicSub(custing->dVD->adj[dest]->ew+*vCurr,1);
			}
			// else {
			// 	atomicAdd(outPutTriangles + uNodes[*uCurr], multiplier);
			// }
		*uCurr += (comp <= 0 && !vmask) || umask;
		*vCurr += (comp >= 0 && !umask) || vmask;
		*workIndex += (comp == 0&& !umask && !vmask) + 1;

		if((*vCurr == vLength) || (*uCurr == uLength)){
			break;
		}
    }
    *triangles -= ((comp == 0) && (*workIndex > *workPerThread) && (found));
  }
}



// u_len < v_len
template <bool uMasked, bool vMasked, bool subtract, bool upd3rdV>
__device__ triangle_t count_trianglesAsymmetric(cuStinger* custing,vertexId_t u, vertexId_t const * const __restrict__ u_nodes, length_t u_len,
    vertexId_t v, vertexId_t const * const __restrict__ v_nodes, length_t v_len, int threads_per_block,
    volatile vertexId_t* __restrict__ firstFound, int tId, triangle_t * const __restrict__ outPutTriangles,
    vertexId_t const * const __restrict__ uMask, vertexId_t const * const __restrict__ vMask, triangle_t multiplier, 
	vertexId_t src, vertexId_t dest)
{
	// Partitioning the work to the multiple thread of a single GPU processor. The threads should get a near equal number of the elements to Tersect - this number will be off by 1.
	int work_per_thread, diag_id;
	workPerThread(u_len, v_len, threads_per_block, tId, &work_per_thread, &diag_id);
	triangle_t triangles = 0;
	int work_index = 0,found=0;
	length_t u_min,u_max,v_min,v_max,u_curr,v_curr;

	firstFound[tId]=0;

	if(work_per_thread>0){
		// For the binary search, we are figuring out the initial poT of search.
		initialize(diag_id, u_len, v_len,&u_min, &u_max,&v_min, &v_max,&found);
    	u_curr = 0; v_curr = 0;

	    bSearch(found, diag_id, u_nodes, v_nodes, &u_len, &u_min, &u_max, &v_min,
        &v_max, &u_curr, &v_curr);

    	int sum = fixStartPoint(u_len, v_len, &u_curr, &v_curr, u_nodes, v_nodes);
    	work_index += sum;
	    if(tId > 0)
	      firstFound[tId-1] = sum;
	    triangles += sum;
	    intersectCountAsymmetric<uMasked, vMasked, subtract, upd3rdV>(custing,
	    	u_len, v_len, u_nodes, v_nodes, &u_curr, &v_curr,
	        &work_index, &work_per_thread, &triangles, firstFound[tId], outPutTriangles, 
	        uMask, vMask, multiplier,src,dest,u,v);
	}
	return triangles;
}





__global__ void deviceBUTwoCUOneTriangles (cuStinger* custing, BatchUpdateData *bud,
    triangle_t * const __restrict__ outPutTriangles, const int threads_per_block,
    const int number_blocks, const int shifter, bool deletion,
    length_t const * const __restrict__ redCU, length_t const * const __restrict__ redBU)
{
	length_t batchsize = *(bud->getBatchSize());
	// Partitioning the work to the multiple thread of a single GPU processor. The threads should get a near equal number of the elements to intersect - this number will be off by no more than one.
	int tx = threadIdx.x;
 	vertexId_t this_mp_start, this_mp_stop;

	length_t *d_off = bud->getOffsets();
	vertexId_t * d_ind = bud->getDst();
	vertexId_t * d_seg = bud->getSrc();

	const int blockSize = blockDim.x;
	workPerBlock(batchsize, &this_mp_start, &this_mp_stop, blockSize);

	__shared__ vertexId_t firstFound[1024];

	length_t adj_offset=tx>>shifter;
	length_t* firstFoundPos=firstFound + (adj_offset<<shifter);
	for (length_t edge = this_mp_start+adj_offset; edge < this_mp_stop; edge+=number_blocks){
		if (bud->getIndDuplicate()[edge]) // this means it's a duplicate edge
			continue;
			
		vertexId_t src = bud->getSrc()[edge];
		vertexId_t dest= bud->getDst()[edge];

		// length_t srcLen= redBU[src];
		// length_t destLen=redCU[dest];
		length_t srcLen= d_off[src+1] - d_off[src];
		length_t destLen=custing->dVD->getUsed()[dest];


		bool avoidCalc = (src == dest) || (srcLen == 0);
		if(avoidCalc)
			continue;

        vertexId_t const * const src_ptr = d_ind + d_off[src];
        vertexId_t const * const src_mask_ptr = bud->getIndDuplicate() + d_off[src];
        vertexId_t const * const dst_ptr = custing->dVD->getAdj()[dest]->dst;

		bool sourceSmaller = (srcLen<destLen);
        vertexId_t small = sourceSmaller? src : dest;
        vertexId_t large = sourceSmaller? dest : src;
        length_t small_len = sourceSmaller? srcLen : destLen;
        length_t large_len = sourceSmaller? destLen : srcLen;

        vertexId_t const * const small_ptr = sourceSmaller? src_ptr : dst_ptr;
        vertexId_t const * const small_mask_ptr = sourceSmaller? src_mask_ptr : NULL;
        vertexId_t const * const large_ptr = sourceSmaller? dst_ptr : src_ptr;
        vertexId_t const * const large_mask_ptr = sourceSmaller? NULL : src_mask_ptr;



		// triangle_t tCount=0;
		triangle_t tCount = (sourceSmaller)?
								count_trianglesAsymmetric<true,false,true,true>(
								custing,small, small_ptr, small_len,
								large,large_ptr, large_len,
								threads_per_block,firstFoundPos,
								tx%threads_per_block, outPutTriangles,
								small_mask_ptr, large_mask_ptr, 1,src,dest):
								count_trianglesAsymmetric<false,true,true,true>(
								custing,small, small_ptr, small_len,
								large,large_ptr, large_len,
								threads_per_block,firstFoundPos,
								tx%threads_per_block, outPutTriangles,
								small_mask_ptr, large_mask_ptr, 1,src,dest)
							;

		atomicSub(outPutTriangles + src, tCount*1);
		atomicSub(outPutTriangles + dest, tCount*1);
		__syncthreads();
	}
}


void callDeviceDifferenceTriangles(cuStinger& custing, BatchUpdate& bu,
    triangle_t * const __restrict__ outPutTriangles, const int threads_per_intersection,
    const int num_intersec_perblock, const int shifter, const int thread_blocks,
    const int blockdim, bool deletion)
{
	dim3 numBlocks(1, 1);
	length_t batchsize = *(bu.getHostBUD()->getBatchSize());
	length_t nv = *(bu.getHostBUD()->getNumVertices());

	numBlocks.x = ceil((float)nv/(float)blockdim);
	length_t* redCU;length_t* redBU;

	numBlocks.x = ceil((float)(batchsize*threads_per_intersection)/(float)blockdim);

	// cout << "The block dim is " << blockdim << " and the number of blocks is  " << numBlocks.x << endl;

	// Calculate all new traingles regardless of repetition
		devicecuStingerNewTriangles<<<numBlocks, blockdim>>>(custing.devicePtr(), bu.getDeviceBUD()->devicePtr(), outPutTriangles, threads_per_intersection,num_intersec_perblock,shifter,deletion, redCU);

	// Calculate triangles formed by ALL new edges
		// deviceBUThreeTriangles<<<numBlocks,blockdim>>>(custing.devicePtr(), bu.getDeviceBUD()->devicePtr(), outPutTriangles, threads_per_intersection,num_intersec_perblock,shifter,deletion,redBU);
	
	// Calculate triangles formed by two new edges
		deviceBUTwoCUOneTriangles<<<numBlocks,blockdim>>>(custing.devicePtr(), bu.getDeviceBUD()->devicePtr(), outPutTriangles, threads_per_intersection,num_intersec_perblock,shifter,deletion,redCU,redBU);
}



