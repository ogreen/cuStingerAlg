#include "hip/hip_runtime.h"


#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <inttypes.h>

#include <math.h>

#include "update.hpp"
#include "cuStinger.hpp"

#include "operators.cuh"

#include "static_k_truss/k_truss.cuh"	

using namespace cuStingerAlgs;

void KTrussOneIteration(cuStinger& custing,
    triangle_t * const __restrict__ outPutTriangles, const int threads_per_block,
    const int number_blocks, const int shifter, const int thread_blocks, const int blockdim, kTrussData* devData);


namespace cuStingerAlgs {

/// TODO - changed hostKatzdata to pointer so that I can try to inherit it in the streaming case.
	



void kTruss::setInitParameters(length_t nv, length_t ne,length_t maxK, int tsp, int nbl, int shifter,int blocks, int  sps){
	hostKTrussData.nv 	= nv;
	hostKTrussData.ne 	= ne;
	hostKTrussData.maxK = maxK;
	hostKTrussData.currK = 3;

	hostKTrussData.tsp 		= tsp;
	hostKTrussData.nbl 		= nbl;
	hostKTrussData.shifter 	= shifter;
	hostKTrussData.blocks 	= blocks;
	hostKTrussData.sps 		=sps;


	if (hostKTrussData.currK>hostKTrussData.maxK){
		cout << "**** The smallest supported TRUSS is k=3 ****" << endl;
	}
}


void kTruss::Init(cuStinger& custing){

	hostKTrussData.isActive 		 =  (int32_t*) allocDeviceArray(hostKTrussData.nv, sizeof(int32_t));
	hostKTrussData.offsetArray	     =  (int32_t*) allocDeviceArray(hostKTrussData.nv+1, sizeof(int32_t));
	hostKTrussData.trianglePerVertex =  (triangle_t*) allocDeviceArray(hostKTrussData.nv, sizeof(triangle_t));
	hostKTrussData.trianglePerEdge	 =  (triangle_t*) allocDeviceArray(hostKTrussData.ne, sizeof(triangle_t));
	hostKTrussData.src				 =  (vertexId_t*) allocDeviceArray(hostKTrussData.ne, sizeof(vertexId_t));
	hostKTrussData.dst	 			 =  (vertexId_t*) allocDeviceArray(hostKTrussData.ne, sizeof(vertexId_t));

	deviceKTrussData = (kTrussData*)allocDeviceArray(1, sizeof(kTrussData));

	cusLB = new cusLoadBalance(custing);

	SyncDeviceWithHost();
	Reset();
}

void kTruss::copyOffsetArrayHost(length_t* hostOffsetArray){
	copyArrayHostToDevice(hostOffsetArray, hostKTrussData.offsetArray, hostKTrussData.nv+1, sizeof(length_t));
}

void kTruss::copyOffsetArrayDevice(length_t* deviceOffsetArray){
	copyArrayDeviceToDevice(deviceOffsetArray, hostKTrussData.offsetArray, hostKTrussData.nv+1, sizeof(length_t));
}

void kTruss::Reset(){
	hostKTrussData.currK 	= 3;
	hostKTrussData.counter 	= 0;
	resetEdgeArray();
	resetVertexArray();

	SyncDeviceWithHost();
}

void kTruss::resetVertexArray(){
	hipMemset((void*)hostKTrussData.trianglePerVertex,0,hostKTrussData.nv*sizeof(int));
}


void kTruss::resetEdgeArray(){
	hipMemset((void*)hostKTrussData.trianglePerEdge,0,hostKTrussData.ne*sizeof(int));
}

void kTruss::Release(){
	delete cusLB;

	freeDeviceArray(hostKTrussData.isActive);
	freeDeviceArray(hostKTrussData.offsetArray);
	freeDeviceArray(hostKTrussData.trianglePerEdge);
	freeDeviceArray(hostKTrussData.trianglePerVertex);

	freeDeviceArray(deviceKTrussData);

}

void kTruss::Run(cuStinger& custing){

	allVinG_TraverseVertices<kTrussOperators::init>(custing,deviceKTrussData);

	Reset();
	hostKTrussData.activeVertices=custing.nv;
	SyncDeviceWithHost();

	while(hostKTrussData.currK  < hostKTrussData.maxK && hostKTrussData.activeVertices>0){


		KTrussOneIteration(custing, hostKTrussData.trianglePerVertex, hostKTrussData.tsp,
				hostKTrussData.nbl,hostKTrussData.shifter,hostKTrussData.blocks, hostKTrussData.sps,
				deviceKTrussData);

		// cout << "Current number of deleted edges is " << hostKTrussData.counter << endl;

		allVinG_TraverseVertices<kTrussOperators::findUnderK>(custing,deviceKTrussData);
		SyncHostWithDevice();
		// cout << "Current number of deleted edges is " << hostKTrussData.counter << endl;
		BatchUpdateData *bud;
		BatchUpdate* bu;
		if(hostKTrussData.counter!=0){
			bud = new BatchUpdateData(hostKTrussData.counter,true,hostKTrussData.nv);
			copyArrayDeviceToHost(hostKTrussData.src,bud->getSrc(),hostKTrussData.counter,sizeof(int));
			copyArrayDeviceToHost(hostKTrussData.dst,bud->getDst(),hostKTrussData.counter,sizeof(int));

			bu = new BatchUpdate(*bud);

			bu->sortDeviceBUD(hostKTrussData.sps);
			// cout << "Hello" << endl;
			// for(int32_t e=0; e<hostKTrussData.counter; e++){
			// 	if(bud->getSrc()[e]> 18772|| bud->getDst()[e] > 18772 )
			// 	printf("Batch update: (#%d) (%d %d)\n", e,bud->getSrc()[e],bud->getDst()[e]);
			// }
			// length_t allocs;
			// custing.edgeInsertions(*bu,allocs);
			custing.edgeDeletionsSorted(*bu);
			delete bu;
			delete bud;

		}

		hostKTrussData.activeVertices=0;
	
		SyncDeviceWithHost();

		allVinG_TraverseVertices<kTrussOperators::countActive>(custing,deviceKTrussData);
		SyncHostWithDevice();

		resetEdgeArray();
		resetVertexArray();


		hostKTrussData.currK++;
		hostKTrussData.counter=0;

		SyncDeviceWithHost();
	}

	
	// hostKTrussData.nActive = hostKTrussData.nv;
	// while(hostKTrussData.nActive  > hostKTrussData.K && hostKTrussData.iteration < hostKTrussData.maxIteration){

	// 	hostKTrussData.alphaI          = pow(hostKTrussData.alpha,hostKTrussData.iteration);
	// 	hostKTrussData.lowerBoundConst = pow(hostKTrussData.alpha,hostKTrussData.iteration+1)/((1.0-hostKTrussData.alpha));
	// 	hostKTrussData.upperBoundConst = pow(hostKTrussData.alpha,hostKTrussData.iteration+1)/((1.0-hostKTrussData.alpha*(double)hostKTrussData.maxDegree));
	// 	hostKTrussData.nActive = 0; // Each iteration the number of active vertices is set to zero.
	
	// 	SyncDeviceWithHost(); // Passing constants to the device.

	// 	allVinG_TraverseVertices<kTrussOperator::initNumPathsPerIteration>(custing,deviceKTrussData);
	// 	allVinA_TraverseEdges_LB<kTrussOperator::updatePathCount>(custing,deviceKTrussData,*cusLB);
	// 	allVinG_TraverseVertices<kTrussOperator::updateKatzAndBounds>(custing,deviceKTrussData);

	// 	SyncHostWithDevice();
	// 	hostKTrussData.iteration++;

	// 	if(isStatic){
	// 		// Swapping pointers.
	// 		ulong_t* temp = hostKTrussData.nPathsCurr; hostKTrussData.nPathsCurr=hostKTrussData.nPathsPrev; hostKTrussData.nPathsPrev=temp;	
	// 	}else{
	// 		hostKTrussData.nPathsPrev = hPathsPtr[hostKTrussData.iteration - 1];
	// 		hostKTrussData.nPathsCurr = hPathsPtr[hostKTrussData.iteration - 0];
	// 	}

	// 	length_t oldActiveCount = hostKTrussData.nActive;
	// 	hostKTrussData.nActive = 0; // Resetting active vertices for sorting operations.

	// 	SyncDeviceWithHost();

	// 	mergesort(hostKTrussData.lowerBoundSort,hostKTrussData.vertexArray,oldActiveCount, greater_t<double>(),context);

	// 	// allVinG_TraverseVertices<kTrussOperator::countActive>(custing,deviceKTrussData);
	// 	allVinA_TraverseVertices<kTrussOperator::countActive>(custing,deviceKTrussData,hostKTrussData.vertexArray,oldActiveCount);

	// 	//allVinA_TraverseVertices<kTrussOperator::printKID>(custing,deviceKTrussData,hostKTrussData.vertexArray, custing.nv);
	// 	SyncHostWithDevice();
	// 	cout << hostKTrussData.nActive << endl;
	// }
	// // cout << "@@ " << hostKTrussData.iteration << " @@" << endl;
	// SyncHostWithDevice();
}

// length_t kTruss::getIterationCount(){
// 	SyncHostWithDevice();
// 	return hostKTrussData.iteration;
// }


}// cuStingerAlgs namespace
