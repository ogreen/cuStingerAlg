

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <inttypes.h>

#include <math.h>

#include "update.hpp"
#include "cuStinger.hpp"

#include "operators.cuh"

#include "static_k_truss/k_truss.cuh"	

using namespace cuStingerAlgs;

void KTrussOneIteration(cuStinger& custing,
    triangle_t * const __restrict__ outPutTriangles, const int threads_per_block,
    const int number_blocks, const int shifter, const int thread_blocks, const int blockdim, kTrussData* devData);


namespace cuStingerAlgs {


void kTruss::setInitParameters(length_t nv, length_t ne,int tsp, int nbl, int shifter,int blocks, int  sps){
	hostKTrussData.nv 	= nv;
	hostKTrussData.ne 	= ne;

	hostKTrussData.tsp 		= tsp;
	hostKTrussData.nbl 		= nbl;
	hostKTrussData.shifter 	= shifter;
	hostKTrussData.blocks 	= blocks;
	hostKTrussData.sps 		=sps;
}


void kTruss::Init(cuStinger& custing){

	hostKTrussData.isActive 		 =  (int32_t*) allocDeviceArray(hostKTrussData.nv, sizeof(int32_t));
	hostKTrussData.offsetArray	     =  (int32_t*) allocDeviceArray(hostKTrussData.nv+1, sizeof(int32_t));
	hostKTrussData.trianglePerVertex =  (triangle_t*) allocDeviceArray(hostKTrussData.nv, sizeof(triangle_t));
	hostKTrussData.trianglePerEdge	 =  (triangle_t*) allocDeviceArray(hostKTrussData.ne, sizeof(triangle_t));
	hostKTrussData.src				 =  (vertexId_t*) allocDeviceArray(hostKTrussData.ne, sizeof(vertexId_t));
	hostKTrussData.dst	 			 =  (vertexId_t*) allocDeviceArray(hostKTrussData.ne, sizeof(vertexId_t));

	deviceKTrussData = (kTrussData*)allocDeviceArray(1, sizeof(kTrussData));

	cusLB = new cusLoadBalance(custing);

	SyncDeviceWithHost();
	Reset();
}

void kTruss::copyOffsetArrayHost(length_t* hostOffsetArray){
	copyArrayHostToDevice(hostOffsetArray, hostKTrussData.offsetArray, hostKTrussData.nv+1, sizeof(length_t));
}

void kTruss::copyOffsetArrayDevice(length_t* deviceOffsetArray){
	copyArrayDeviceToDevice(deviceOffsetArray, hostKTrussData.offsetArray, hostKTrussData.nv+1, sizeof(length_t));
}

void kTruss::Reset(){
	hostKTrussData.counter 	= 0;
	hostKTrussData.ne_remaining	= hostKTrussData.ne;

	resetEdgeArray();
	resetVertexArray();

	SyncDeviceWithHost();
}

void kTruss::resetVertexArray(){
	hipMemset((void*)hostKTrussData.trianglePerVertex,0,hostKTrussData.nv*sizeof(int));
}


void kTruss::resetEdgeArray(){
	hipMemset((void*)hostKTrussData.trianglePerEdge,0,hostKTrussData.ne*sizeof(int));
}

void kTruss::Release(){
	delete cusLB;

	freeDeviceArray(hostKTrussData.isActive);
	freeDeviceArray(hostKTrussData.offsetArray);
	freeDeviceArray(hostKTrussData.trianglePerEdge);
	freeDeviceArray(hostKTrussData.trianglePerVertex);

	freeDeviceArray(deviceKTrussData);

}

void kTruss::Run(cuStinger& custing){

	hostKTrussData.maxK = 3;SyncDeviceWithHost();

	while(1){
		bool exitOnFirstIteration=false;
		bool more = findTrussOfK(custing,exitOnFirstIteration);
		if(more==false && exitOnFirstIteration){
			hostKTrussData.maxK--; SyncDeviceWithHost();
			break;
		}
		hostKTrussData.maxK++; SyncDeviceWithHost();
	}
	// cout << "Found the maximal KTruss at : " << hostKTrussData.maxK << endl;
}

void kTruss::RunForK(cuStinger& custing,int maxK){

	hostKTrussData.maxK = maxK;SyncDeviceWithHost();

	bool exitOnFirstIteration;
	findTrussOfK(custing,exitOnFirstIteration);
}


bool kTruss::findTrussOfK(cuStinger& custing, bool& stop){

	allVinG_TraverseVertices<kTrussOperators::init>(custing,deviceKTrussData);

	// Reset();
	resetEdgeArray();
	resetVertexArray();

	hostKTrussData.counter 	= 0;
	hostKTrussData.activeVertices=custing.nv;
	SyncDeviceWithHost();
	int sumDeletedEdges=0;
	stop=true;

	while(hostKTrussData.activeVertices>0){

		KTrussOneIteration(custing, hostKTrussData.trianglePerVertex, hostKTrussData.tsp,
				hostKTrussData.nbl,hostKTrussData.shifter,hostKTrussData.blocks, hostKTrussData.sps,
				deviceKTrussData);

		allVinG_TraverseVertices<kTrussOperators::findUnderK>(custing,deviceKTrussData);
		SyncHostWithDevice();
		// cout << "Current number of deleted edges is " << hostKTrussData.counter << endl;
		sumDeletedEdges+=hostKTrussData.counter;
		if(hostKTrussData.counter==hostKTrussData.ne_remaining){
			stop = true;
			return false;
		}
		if(hostKTrussData.counter!=0){
			BatchUpdateData *bud;
			BatchUpdate* bu;
			bud = new BatchUpdateData(hostKTrussData.counter,true,hostKTrussData.nv);
			copyArrayDeviceToHost(hostKTrussData.src,bud->getSrc(),hostKTrussData.counter,sizeof(int));
			copyArrayDeviceToHost(hostKTrussData.dst,bud->getDst(),hostKTrussData.counter,sizeof(int));
			bu = new BatchUpdate(*bud);

			bu->sortDeviceBUD(hostKTrussData.sps);
			custing.edgeDeletionsSorted(*bu);
			delete bu;
			delete bud;
		}
		else{
			// cout << "The maxK is                  : " << hostKTrussData.maxK << endl;
			// cout << "This is the first iteration  : " << stop << endl;
			// cout << "The number of delete edges   : " << sumDeletedEdges <<  endl;
			// cout << "The number of leftover edges : " << hostKTrussData.ne_remaining<< endl;

			return false;
		}
		hostKTrussData.ne_remaining-=hostKTrussData.counter;

		hostKTrussData.activeVertices=0;
	
		SyncDeviceWithHost();

		allVinG_TraverseVertices<kTrussOperators::countActive>(custing,deviceKTrussData);
		SyncHostWithDevice();

		resetEdgeArray();
		resetVertexArray();
		
		hostKTrussData.counter=0;

		SyncDeviceWithHost();
		stop=false;
	}

	return true;
}



}// cuStingerAlgs namespace
